#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <vector>

using std::begin;
using std::copy;
using std::cout;
using std::endl;
using std::end;
using std::generate;
using std::vector;


__global__ void add(int* out, int* a, int* b, int n) {
    out[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
    /* for (int i = 0; i < 10; i++) {
        out[i] = a[i] + b[i];
    }*/
}


int main() {
    int* a, * b, * out;
    int N = 100;
    int size = sizeof(int) * N;
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    out = (int*)malloc(size);
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = N;
        out[i] = 0;
    }

    int* d_a, * d_b, * d_out;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);
    cout << " COMPLETED SUCCESSFULLY\n";

    // blocks, threads per block. 1024 threads per block
    // add <<<N/256 + 1, 256>>
    add << <N, 1>> > (d_out, d_a, d_b, N);
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
    // not &out
    
    for (int i = 0; i < N; i++) {
        cout << out[i] << endl;
    }
    cout << " COMPLETED SUCCESSFULLY\n";

    //    //// Cleanup after kernel execution
    hipFree(d_a);    hipFree(d_b);    hipFree(d_out);
    free(a);    free(b);
    return 0;

}
